
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_fp16.h>

// CUDA kernel to perform matrix multiplication
__global__ void matrixMul(float *a, float *b, float *c, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        half sum = 0;
        for (int i = 0; i < n; ++i) {
            half a_h = __float2half(a[row * n + i]);
            half b_h = __float2half(b[i * n + col]);
            sum = __hfma(a_h, b_h, sum);
        }
        c[row * n + col] = __half2float(sum);
    }
}

int main(int argc, char** argv) {
    if (argc != 2){
        printf("Missing input arg\n");
        return -1;
    }

    int N = atoi(argv[1]);
    float *a, *b, *c; // Host matrices
    float *d_a, *d_b, *d_c; // Device matrices

    // Allocate memory for host matrices
    a = (float*)malloc(N * N * sizeof(float));
    b = (float*)malloc(N * N * sizeof(float));
    c = (float*)malloc(N * N * sizeof(float));

    // Initialize host matrices
    for (int i = 0; i < N * N; ++i) {
        a[i] = i;
        b[i] = i;
    }

    // Allocate memory for device matrices
    hipMalloc((void**)&d_a, N * N * sizeof(float));
    hipMalloc((void**)&d_b, N * N * sizeof(float));
    hipMalloc((void**)&d_c, N * N * sizeof(float));

    // Copy host matrices to device
    hipMemcpy(d_a, a, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * N * sizeof(float), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 dimGrid((N + 15) / 16, (N + 15) / 16);
    dim3 dimBlock(16, 16);

    //create GPU timing events for timing the GPU
    hipEvent_t st2, et2;
    hipEventCreate(&st2);
    hipEventCreate(&et2);        
     
    hipEventRecord(st2);

    // Launch kernel
    matrixMul<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, N);

    hipEventRecord(et2);

    //host waits until et2 has occured     
    hipEventSynchronize(et2);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, st2, et2);

    printf("N=%d Kernel time: %f ns\n", N, 1000000 * milliseconds);

    // Copy result from device to host
    hipMemcpy(c, d_c, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // Display result
    printf("Result Matrix:\n");
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            printf("%f ", c[i * N + j]);
        }
        printf("\n");
    }

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free host memory
    free(a);
    free(b);
    free(c);

    return 0;
}
